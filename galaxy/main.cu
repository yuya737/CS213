#include "hip/hip_runtime.h"
#include <math.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h>

#include <SDL.h>

#include "gui.h"

// Time step size
#define DT 0.075

// Gravitational constant
#define G 100

// Threads per block
#define THREADS_PER_BLOCK 32

// This struct holds data for a single star
typedef struct star {
    float x_position;
    float y_position;
    float x_velocity;
    float y_velocity;
    float mass;
} star_t;

// Generate a random float in a given range
float drand(float min, float max) {
    return ((float)rand() / RAND_MAX) * (max - min) + min;
}

// Compute the radius of a star based on its mass
float star_radius(float mass) {
    return sqrt(mass);
}

__global__ void updatePosition(star_t* starsGPU, int num_stars){
    // Calculate the index of starsGPU this should handle. If it exceeds the bounds, return
    int i = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    if (i >= num_stars) return;

    starsGPU[i].x_position += starsGPU[i].x_velocity * DT;
    starsGPU[i].y_position += starsGPU[i].y_velocity * DT;
    for(int j=0; j<num_stars; j++) {
        // Don't compute the force of a star on itself
        if(i == j) continue;

        // Compute the distance between the two stars in each dimension
        float x_diff = starsGPU[i].x_position - starsGPU[j].x_position;
        float y_diff = starsGPU[i].y_position - starsGPU[j].y_position;

        // Compute the magnitude of the distance vector
        float dist = sqrt(x_diff * x_diff + y_diff * y_diff);

        // Normalize the distance vector components
        x_diff /= dist;
        y_diff /= dist;

        // Keep a minimum distance, otherwise we get 
        float combined_radius = sqrt(starsGPU[i].mass) + sqrt(starsGPU[j].mass);
        if(dist < combined_radius) {
            dist = combined_radius;
        }

        // Compute the x and y accelerations
        float x_acceleration = -x_diff * G * starsGPU[j].mass / (dist * dist);
        float y_acceleration = -y_diff * G * starsGPU[j].mass / (dist * dist);

        // Update the star velocity
        starsGPU[i].x_velocity += x_acceleration * DT;
        starsGPU[i].y_velocity += y_acceleration * DT;

        // Handle edge collisiosn
        if(starsGPU[i].x_position < 0 && starsGPU[i].x_velocity < 0) starsGPU[i].x_velocity *= -0.5;
        if(starsGPU[i].x_position >= SCREEN_WIDTH && starsGPU[i].x_velocity > 0) starsGPU[i].x_velocity *= -0.5;
        if(starsGPU[i].y_position < 0 && starsGPU[i].y_velocity < 0) starsGPU[i].y_velocity *= -0.5;
        if(starsGPU[i].y_position >= SCREEN_HEIGHT && starsGPU[i].y_velocity > 0) starsGPU[i].y_velocity *= -0.5;
    }
}

int main(int argc, char** argv) {
    // Initialize the graphical interface
    gui_init();

    // Run as long as this is true
    bool running = true;

    // Is the mouse currently clicked?
    bool clicked = false;

    // This will hold our array of stars for CPU and GPU
    star_t* stars = NULL;
    star_t* starsGPU = NULL;
    int num_stars = 0;
    // Keep a count of the previous count of stars, so that we only copy if the number of stars increase
    int prevStarCount = num_stars;

    // Start main loop
    while(running) {
        // Check for events
        SDL_Event event;
        while(SDL_PollEvent(&event) == 1) {
            // If the event is a quit event, then leave the loop
            if(event.type == SDL_QUIT) running = false;
        }

        // Get the current mouse state
        int mouse_x, mouse_y;
        uint32_t mouse_state = SDL_GetMouseState(&mouse_x, &mouse_y);

        // Is the mouse pressed?
        if(mouse_state & SDL_BUTTON(SDL_BUTTON_LEFT)) {
            // Is this the beginning of a mouse click?
            if(!clicked) {
                clicked = true;
                stars = (star_t*)realloc(stars, (num_stars + 1) * sizeof(star_t));
                stars[num_stars].x_position = mouse_x + drand(-1, 1);
                stars[num_stars].y_position = mouse_y + drand(-1, 1);
                stars[num_stars].x_velocity = 0;
                stars[num_stars].y_velocity = 0;
                // Generate a random mass skewed toward small sizes
                stars[num_stars].mass = drand(0, 1) * drand(0, 1) * 50;
                num_stars++;
            }
        } else {
            // The mouse click is finished
            clicked = false;
        }

        // Draw stars
        for(int i=0; i<num_stars; i++) {
            color_t color = {255, 255, 255, 255};
            gui_draw_circle(stars[i].x_position, stars[i].y_position, star_radius(stars[i].mass), color);
        }

        // If there are different number of stars (i.e. stars are added) then, free starsGPU list, allocate and copy stars to starsGPU
        if (num_stars != prevStarCount){
            if (starsGPU != NULL) hipFree(starsGPU);
            hipMalloc((void**) &starsGPU, num_stars*sizeof(star_t));
            if (hipMemcpy(starsGPU, stars, sizeof(star_t)*num_stars, hipMemcpyHostToDevice) != hipSuccess){
                fprintf(stderr, "Failed to copy stars to starsGPU\n");
            }
        }

        // Calculate the number of blocks that should be run
        int blocks = (num_stars + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        // Update stars, run calculations on GPU, wait for completion and copy back to host to display
        updatePosition<<<blocks, THREADS_PER_BLOCK>>>(starsGPU, num_stars);
        if(hipDeviceSynchronize() != hipSuccess) {
            fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));         
        }
        if (hipMemcpy(stars, starsGPU, sizeof(star_t)*num_stars, hipMemcpyDeviceToHost) != hipSuccess){
            fprintf(stderr, "Failed to copy starsGPU to stars\n");
        }
        // update prevStarCount
        prevStarCount = num_stars;

        // Display the rendered image
        gui_update_display();

        // Fade out the rendered image to leave movement trails
        gui_fade(0.85);
    }

    // Free the stars array
    free(stars);

    // Free the starsGPU array
    hipFree(starsGPU);
    // Clean up the graphical interface
    gui_shutdown();

    return 0;
}
